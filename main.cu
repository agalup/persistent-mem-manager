#include "hip/hip_runtime.h"
#include "pmm.cu"
using namespace std;

int main(int argc, char *argv[]){

    //size_t instant_size = 8 * 1024ULL * 1024ULL * 1024ULL;
    size_t instant_size = 4 * 1024ULL * 1024ULL * 1024ULL;
    int size_to_alloc = 4;
    int iteration_num = 1;

    int turn_on = 1;
    /*if (argc > 1){
        turn_on = atoi(argv[1]);
    }*/
    if (argc > 1){
        size_to_alloc = atoi(argv[1]);
    }
    if (argc > 2){
        iteration_num = atoi(argv[2]);
    }
    if (argc > 3){
        instant_size = atoi(argv[3]);
    }

   // int* app_launch = (int*)malloc
    hipDeviceProp_t deviceProp;
    GUARD_CU(hipGetDeviceProperties(&deviceProp, 0));
    int SMs = deviceProp.multiProcessorCount;
    printf("max block number %d\n", SMs);
    printf("instant size %ld\n", instant_size);
    int size = SMs - 1;

    int* sm_app             = (int*)malloc(sizeof(int)*size);
    int* sm_mm              = (int*)malloc(sizeof(int)*size);
    int* sm_gc              = (int*)malloc(sizeof(int)*size);
    int* allocs_size        = (int*)malloc(sizeof(int)*size);
    float* malloc_sync      = (float*)malloc(sizeof(float)*size);
    float* malloc_per_sec   = (float*)malloc(sizeof(float)*size);
    float* free_sync        = (float*)malloc(sizeof(float)*size);
    float* free_per_sec     = (float*)malloc(sizeof(float)*size);
    //float* app_sync        = (float*)malloc(sizeof(float)*size);
    //float* uni_req_num     = (float*)malloc(sizeof(float)*size);
    
    pmm_init(turn_on, size_to_alloc, &instant_size, iteration_num, SMs, 
            sm_app, sm_mm, sm_gc, allocs_size, malloc_sync, malloc_per_sec, 
            free_sync, free_per_sec);

    GUARD_CU(hipDeviceReset());
    GUARD_CU(hipPeekAtLastError());
/*
    perf_alloc(size_to_alloc, &instant_size, iteration_num, SMs, 
            app_sync, uni_req_num, turn_on);*/

    printf("DONE!\n");
    return 0;
}

