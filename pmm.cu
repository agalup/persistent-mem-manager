#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <cassert>
#include <algorithm>

#include "device/Ouroboros_impl.cuh"
#include "device/MemoryInitialization.cuh"
#include "InstanceDefinitions.cuh"
#include "PerformanceMeasure.cuh"
#include "Utility.h"
#include "hip/hip_runtime.h"
#include "pmm-utils.cuh"

using namespace std;

//#define DEBUG
#ifdef DEBUG
#define debug(a...) printf(a)
#else
#define debug(a...)
#endif

/*
#ifndef HALLOC__
    #ifndef OUROBOROS__
        #define OUROBOROS__
    #endif
#endif*/

#ifdef HALLOC__
#include "Instance.cuh"
#endif

extern "C"{

#ifdef OUROBOROS__
    //Ouroboros initialization
    #define MemoryManagerType OuroPQ
#endif
#ifdef HALLOC__
    //Halloc initialization
    #define MemoryManagerType MemoryManagerHalloc
#endif

__global__
void mem_free(volatile int** d_memory, 
#ifdef OUROBOROS__
              //OuroPQ* mm,
              MemoryManagerType* mm, 
#else 
    #ifdef HALLOC__
              //MemoryManagerHalloc mm,
              MemoryManagerType mm, 
    #endif
#endif
              volatile int* requests_num
        ){
    int thid = blockDim.x * blockIdx.x + threadIdx.x;
    if (thid >= requests_num[0]){
        return;
    }
#ifdef OUROBOROS__
            if (d_memory[thid])
                mm->free((void*)d_memory[thid]);
#else 
    #ifdef HALLOC__
            if (d_memory[thid])
                mm.free((void*)d_memory[thid]);
    #endif
#endif
}

//producer
__global__
void mem_manager(volatile int* exit_signal, 
                volatile int* requests_number, 
                volatile int* request_iter,
                volatile int* request_signal, 
                volatile int* request_ids, 
#ifdef OUROBOROS__
              //OuroPQ* mm,
              MemoryManagerType* mm, 
#else 
    #ifdef HALLOC__
              //MemoryManagerHalloc mm,
              MemoryManagerType mm, 
    #endif
#endif
                volatile int** d_memory,
                volatile int* request_mem_size,
                volatile int* lock, 
                int turn_on){
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (! exit_signal[0]){
        for (int request_id=thid; request_id<requests_number[0]; 
                request_id += blockDim.x*gridDim.x){

            if (request_signal[request_id] == 1){

                // SEMAPHORE
                acquire_semaphore((int*)lock, request_id);
                debug("mm: request recieved %d\n", request_id); 
                int req_id = atomicAdd((int*)&request_iter[0], 1);
                request_ids[request_id] = req_id;

                if (turn_on){
                    d_memory[req_id] = reinterpret_cast<volatile int*>
#ifdef HALLOC__
                        (mm.malloc(request_mem_size[request_id]));
#else
#ifdef OUROBOROS__
                        (mm->malloc(request_mem_size[request_id]));
#endif
#endif
                    if (!exit_signal[0])
                        assert(d_memory[req_id]);
                }

                // SIGNAL update
                atomicExch((int*)&request_signal[request_id], 2);

                release_semaphore((int*)lock, request_id);
                // SEMAPHORE

                debug("mm: request done %d\n", request_id);
            }
        }
    }
}

//consumer
__global__
void app(volatile int* exit_signal,
         volatile int** d_memory, 
         volatile int* request_signal, 
         volatile int* request_mem_size,
         volatile int* request_id, 
         volatile int* exit_counter, 
         volatile int* lock,
         int size_to_alloc,
         int turn_on){
    int thid = blockDim.x * blockIdx.x + threadIdx.x;

    // SEMAPHORE
    acquire_semaphore((int*)lock, thid);
    request_mem_size[thid] = size_to_alloc;
    request_id[thid] = -1;
    int req_id = -1;
    // SIGNAL update
    atomicExch((int*)&request_signal[thid], 1);
    __threadfence();
    release_semaphore((int*)lock, thid);
    // SEMAPHORE
    
    // wait for success
    while (!exit_signal[0]){
        __threadfence();
        if (request_signal[thid] == 2){
            
            // SEMAPHORE
            acquire_semaphore((int*)lock, thid);
            req_id = request_id[thid];
            if (req_id >= 0 && turn_on && !exit_signal[0]) {
                assert(d_memory[req_id]);
                d_memory[req_id][0] = thid;
            }
            request_signal[thid] = 0;
            __threadfence();
            debug("app: request %d success\n", thid);
            release_semaphore((int*)lock, thid);
            // SEMAPHORE
        
            break;
        }
    }
    atomicAdd((int*)&exit_counter[0], 1);
}

void pmm_init(int turn_on, int size_to_alloc, size_t instant_size, size_t num_iterations, 
            int SMs, int* sm_app, int* sm_mm, int* allocs, float* app_launch, 
            float* app_finish, float* app_sync, float* uni_req_num){

#ifdef OUROBOROS__
    //Ouroboros initialization
    MemoryManagerType memory_manager;
    memory_manager.initialize(instant_size);
#else
#ifdef HALLOC__
    //Halloc initialization
    //size_t instantitation_size = 2048ULL * 1024ULL * 1024ULL;
    MemoryManagerType memory_manager(instant_size);
#endif
#endif
 
    GUARD_CU(hipPeekAtLastError());
    //Creat two asynchronous streams which may run concurrently with the default stream 0.
    //The streams are not synchronized with the default stream.
    hipStream_t mm_stream, app_stream;
    GUARD_CU(hipStreamCreateWithFlags( &mm_stream, hipStreamNonBlocking));
    GUARD_CU(hipPeekAtLastError());
    GUARD_CU(hipStreamCreateWithFlags(&app_stream, hipStreamNonBlocking));
    GUARD_CU(hipPeekAtLastError());
    
    int* exit_signal;
    GUARD_CU(hipMallocManaged(&exit_signal, sizeof(int32_t)));
    GUARD_CU(hipPeekAtLastError());

    int* exit_counter;
    GUARD_CU(hipMallocManaged(&exit_counter, sizeof(uint32_t)));
    GUARD_CU(hipPeekAtLastError());
    
    int block_size = 1024;

    printf("size to alloc per thread %d, num iterations %d\n", size_to_alloc, num_iterations);
    
    std::cout << "\t\t#allocs\t\t" << "#sm app\t\t" << "#sm mm\t\t" << "#req per sec\t\t" << "app launch\t" << "app finished\t" << "app finish sync\n";

    for (int mm_grid_size = 1; mm_grid_size < SMs; ++mm_grid_size){

        int app_grid_size = SMs - mm_grid_size;
        int requests_num{app_grid_size*block_size};

        //output
        sm_app[mm_grid_size - 1] = app_grid_size;
        sm_mm [mm_grid_size - 1] = mm_grid_size;
        allocs[mm_grid_size - 1] = requests_num;

        //Timing variables
        PerfMeasure timing_app, timing_mm, timing_total, timing_total_sync;

        for (int iteration = 0; iteration < num_iterations; ++iteration){

            *exit_signal = 0;
            *exit_counter = 0;

            RequestType requests;
            requests.init(requests_num);
            requests.memset();

            timing_mm.startMeasurement();
            //Run presistent kernel (Memory Manager)
            mem_manager<<<mm_grid_size, block_size, 0, mm_stream>>>(exit_signal,
                    requests.requests_number, 
                    requests.request_iter, 
                    requests.request_signal, 
                    requests.request_id,
#ifdef OUROBOROS__
                    memory_manager.getDeviceMemoryManager(),
#else
#ifdef HALLOC__
                    memory_manager,
#endif
#endif
                    requests.d_memory,
                    requests.request_mem_size,
                    requests.lock, 
                    turn_on);
            timing_mm.stopMeasurement();
            GUARD_CU(hipPeekAtLastError());

            timing_app.startMeasurement();
            timing_total.startMeasurement();
            timing_total_sync.startMeasurement();
            //Run application
            app<<<app_grid_size, block_size, 0, app_stream>>>(exit_signal, 
                    requests.d_memory, 
                    requests.request_signal, 
                    requests.request_mem_size, 
                    requests.request_id, 
                    exit_counter, 
                    requests.lock,
                    size_to_alloc,
                    turn_on);
            timing_app.stopMeasurement();
            GUARD_CU(hipPeekAtLastError());

            // Check results
            int old_counter = -1;
            long long iter = 0;
            long long iter2 = 0;
            long long iter_mean = 0;
            long long  time_limit = 100000000;
            while (iter2 < time_limit){
                if (exit_counter[0] == block_size*app_grid_size){
                    timing_total.stopMeasurement();
                    *exit_signal = 1;
                    GUARD_CU(hipDeviceSynchronize());
                    GUARD_CU(hipPeekAtLastError());
                    timing_total_sync.stopMeasurement();
                    if (turn_on){
                        test1<<<app_grid_size, block_size, 0, app_stream>>>(requests.d_memory);
                        GUARD_CU(hipDeviceSynchronize());
                        GUARD_CU(hipPeekAtLastError());
                        mem_test((int**)requests.d_memory, requests_num, app_grid_size, block_size, mm_stream);
                    }
                    break;
                }else{
                    if (exit_counter[0] != old_counter){
                        old_counter = exit_counter[0];

                        ++iter;
                        iter_mean += iter2;
                        iter2 = 0;

                        if (iter%1000 == 0){
                            //debug("no break, exit_counter = %d\n", exit_counter[0]);
                            //printf("no break, exit_counter = %d, change after %d iterations\n", exit_counter[0],\
                            iter_mean/iter);
                        }
                    }
                    ++iter2;
                }
                if (iter2 >= time_limit){
                    printf("time limit exceed, break\n");
                    *exit_signal = 1;
                }
            }

            if (iter != 0)
                iter_mean /= iter;
         
            //printf("new change each %d iterations\n", iter_mean);
            GUARD_CU(hipPeekAtLastError());
            //Deallocate device memory
            mem_free<<<app_grid_size, block_size, 0, app_stream>>>(
                    requests.d_memory, 
#ifdef OUROBOROS__
                    memory_manager.getDeviceMemoryManager(),
#else
#ifdef HALLOC__
                    memory_manager,
#endif
#endif
                    requests.requests_number);

            GUARD_CU(hipPeekAtLastError());
            requests.free();
            GUARD_CU(hipPeekAtLastError());

        }
        // Output
        auto app_time = timing_app.generateResult();
        auto total_time = timing_total.generateResult();
        auto total_sync_time = timing_total_sync.generateResult();
        app_launch[mm_grid_size - 1] = (app_time.mean_);
        app_finish[mm_grid_size - 1] = (total_time.mean_);
        app_sync  [mm_grid_size - 1] = (total_sync_time.mean_);
        // The number of requests done per a second
        uni_req_num[mm_grid_size - 1] = (requests_num * 1000.0)/total_sync_time.mean_;

        printf("\t\t%d\t\t| %d\t\t| %d\t\t| %.2lf\t\t| %.2lf\t\t| %.2lf\t\t| %.2lf\n", 
                requests_num, app_grid_size, mm_grid_size, uni_req_num[mm_grid_size - 1], app_time.mean_,
                total_time.mean_, total_sync_time.mean_);
    }

    GUARD_CU(hipStreamSynchronize(mm_stream));
    GUARD_CU(hipStreamSynchronize(app_stream));
    GUARD_CU(hipPeekAtLastError());
}

}
